#include "hip/hip_runtime.h"
//
//  CCFJ-py
//  A Python Package for seismic ambient noise cross-correlation and Frequency-Bessel Transform Method
//
//  GNU General Public License, Version 3, 29 June 2007
//
//  Copyright © 2021 Xiaofei Chen Research Group,
//  Department of Earth and Space Sciences,
//  Southern University of Science and Technology, China.
//
#include "cuda_helper.h"
#include "FJgpu.hh"

using namespace std;

__device__ double bessj0( double x )
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate Bessel fun//ction of first kind and order  */
/*          0 at input x                                      */
/*------------------------------------------------------------*/
{
   double ax,z;
   double xx,y,ans,ans1,ans2;

   if ((ax=fabs(x)) < 8.0) {
      y=x*x;
      ans1=57568490574.0+y*(-13362590354.0+y*(651619640.7
         +y*(-11214424.18+y*(77392.33017+y*(-184.9052456)))));
      ans2=57568490411.0+y*(1029532985.0+y*(9494680.718
         +y*(59272.64853+y*(267.8532712+y*1.0))));
      ans=ans1/ans2;
   } else {
      z=8.0/ax;
      y=z*z;
      xx=ax-0.785398164;
      ans1=1.0+y*(-0.1098628627e-2+y*(0.2734510407e-4
         +y*(-0.2073370639e-5+y*0.2093887211e-6)));
      ans2 = -0.1562499995e-1+y*(0.1430488765e-3
         +y*(-0.6911147651e-5+y*(0.7621095161e-6
         -y*0.934935152e-7)));
      ans=sqrt(0.636619772/ax)*(cos(xx)*ans1-z*sin(xx)*ans2);
   }
   return ans;
}

__device__ double bessy0( double x )
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate Bessel function of second kind and order */
/*          0 at input x.                                     */
/*------------------------------------------------------------*/
{
   double z;
   double xx,y,ans,ans1,ans2;

   if (x < 8.0) {
      y=x*x;
      ans1 = -2957821389.0+y*(7062834065.0+y*(-512359803.6
         +y*(10879881.29+y*(-86327.92757+y*228.4622733))));
      ans2=40076544269.0+y*(745249964.8+y*(7189466.438
         +y*(47447.26470+y*(226.1030244+y*1.0))));
      ans=(ans1/ans2)+0.636619772*bessj0(x)*log(x);
   } else {
      z=8.0/x;
      y=z*z;
      xx=x-0.785398164;
      ans1=1.0+y*(-0.1098628627e-2+y*(0.2734510407e-4
         +y*(-0.2073370639e-5+y*0.2093887211e-6)));
      ans2 = -0.1562499995e-1+y*(0.1430488765e-3
         +y*(-0.6911147651e-5+y*(0.7621095161e-6
         +y*(-0.934945152e-7))));
      ans=sqrt(0.636619772/x)*(sin(xx)*ans1+z*cos(xx)*ans2);
   }
   return ans;
}

__device__ double STVH0(double X) {
/*      =============================================
!       Purpose: Compute Struve function H0(x)
!       Input :  x   --- Argument of H0(x) ( x ò 0 )
!       Output:  SH0 --- H0(x)
!       ============================================= */
        double A0,BY0,P0,PI,Q0,R,S,T,T2,TA0;
	int K, KM;
        double SH0;
	    
	PI=3.141592653589793;
        S=1.0;
        R=1.0;
        if (X <= 20.0) {
           A0=2.0*X/PI;
           for (K=1; K<61; K++) {
              R=-R*X/(2.0*K+1.0)*X/(2.0*K+1.0);
              S=S+R;
              if (fabs(R) < fabs(S)*1.0e-12) goto e15;
           }
e15:       SH0=A0*S;
        }
        else {
           KM=int(0.5*(X+1.0));
           if (X >= 50.0) KM=25;
           for (K=1; K<=KM; K++) {
			  //R=-R*pow((2.0*K-1.0)/X,2);
			  R = -R*(2.0*K-1.0)*(2.0*K-1.0)/X/X;
              S=S+R;
              if (fabs(R) < fabs(S)*1.0e-12) goto e25;
           }
e25:       T=4.0/X;
           T2=T*T;
           P0=((((-.37043e-5*T2+.173565e-4)*T2-.487613e-4)*T2+.17343e-3)*T2-0.1753062e-2)*T2+.3989422793;
           Q0=T*(((((.32312e-5*T2-0.142078e-4)*T2+0.342468e-4)*T2-0.869791e-4)*T2+0.4564324e-3)*T2-0.0124669441);
           TA0=X-0.25*PI;
           BY0=2.0/sqrt(X)*(P0*sin(TA0)+Q0*cos(TA0));
           SH0=2.0/(PI*X)*S+BY0;
        }
        return SH0;
}

__device__ double STVH1(double X) {
/*      =============================================
!       Purpose: Compute Struve function H1(x)
!       Input :  x   --- Argument of H1(x) ( x ò 0 )
!       Output:  SH1 --- H1(x)
!       ============================================= */
        double A0,BY1,P1,PI,Q1,R,S,T,T2,TA1;
	int K, KM;
        double SH1;

        PI=3.141592653589793;
        R=1.0;
        if (X <= 20.0) {
           S=0.0;
           A0=-2.0/PI;
           for (K=1; K<=60; K++) {
              R=-R*X*X/(4.0*K*K-1.0);
              S=S+R;
              if (fabs(R) < fabs(S)*1.0e-12) goto e15;
           }
e15:       SH1=A0*S;
        }
        else {
           S=1.0;
           KM=int(0.5*X);
           if (X > 50.0)  KM=25;
           for (K=1; K<=KM; K++) {
              R=-R*(4.0*K*K-1.0)/(X*X);
              S=S+R;
              if (fabs(R) < fabs(S)*1.0e-12) goto e25;
           }
e25:       T=4.0/X;
           T2=T*T;
           P1=((((0.42414e-5*T2-0.20092e-4)*T2+0.580759e-4)*T2-0.223203e-3)*T2+0.29218256e-2)*T2+0.3989422819;
           Q1=T*(((((-0.36594e-5*T2+0.1622e-4)*T2-0.398708e-4)*T2+0.1064741e-3)*T2-0.63904e-3)*T2+0.0374008364);
           TA1=X-0.75*PI;
           BY1=2.0/sqrt(X)*(P1*sin(TA1)+Q1*cos(TA1));
           SH1=2.0/PI*(1.0+S/(X*X))+BY1;
        }
        return SH1;
}

__global__ void trap_J(float *U_f, float *r, float *out, float *c, float *f, int nr, int nc, int nf) {
	bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k, fl, cl, r1, r2, g1, g2;
	float dr0;
	int indx_d;
	if ((gtidx >= nf) || (gtidy >= nc))
		validr = false;
	if (validr) {
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl / cl;
		kernel = 0.0;
		for (int i = 1; i < nr; i++) {
			indx_d = gtidx + i * nf;
			//g1 = U_f[indx_d-1];
			g1 = U_f[indx_d - nf];
			g2 = U_f[indx_d];
			r1 = r[i - 1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1), 0.1);
			kernel += (g1*j0(k*r1)*r1+g2*j0(k*r2)*r2)*dr0/2;
		}
		out[indx] = kernel;
	}
}

__global__ void trap_Y(float *U_f, float *r, float *out, float *c, float *f, int nr, int nc, int nf) {
	bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k, fl, cl, r1, r2, g1, g2;
	float dr0;
	int indx_d;
	if ((gtidx >= nf) || (gtidy >= nc))
		validr = false;
	if (validr) {
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl / cl;
		kernel = 0.0;
		for (int i = 1; i < nr; i++) {
			indx_d = gtidx + i * nf;
			//g1 = U_f[indx_d-1];
			g1 = U_f[indx_d - nf];
			g2 = U_f[indx_d];
			r1 = r[i - 1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1), 0.1);
			kernel += (g1*bessy0(k*r1)*r1+g2*(bessy0(k*r2))*r2)*dr0/2;
		}
		out[indx] = kernel;
	}
}


__global__ void integral_J(float *U_f, float *r, float *out, float *c, float *f, int nr, int nc, int nf) {
	bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k, fl, cl, r1, r2, g1, g2, a, b;
	float dr0, B01, B02;
	int indx_d;
	if ((gtidx >= nf) || (gtidy >= nc))
		validr = false;
	if (validr) {
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl / cl;
		kernel = 0.0;
		for (int i = 1; i < nr; i++) {
			indx_d = gtidx + i * nf;
			//g1 = U_f[indx_d-1];
			g1 = U_f[indx_d - nf];
			g2 = U_f[indx_d];
			r1 = r[i - 1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1), 0.1);
			a = g1 - r1 * (g2 - g1) / dr0;
			b = (g2 - g1) / dr0;
			kernel += a * (r2*j1(k*r2) - r1 * j1(k*r1)) / k;
			kernel += b * (r2*r2*j1(k*r2) - r1 * r1*j1(k*r1)) / k;
			kernel += b * (r2*j0(k*r2) - r1 * j0(k*r1)) / k / k;
			/* B01 = 0.0; */
			/* B0n = floorf(floorf(B01 / 2 + 10)*1.5); */
			/* for (int j = 0; j < B0n; j++) */
			/* 	B01 += 2*jn(j * 2 + 1, k*r1); */
			/* B02 = 0.0; */
			/* B0n = floorf(floorf(B02 / 2 + 10)*1.5); */
			/* for (int j = 0; j < B0n; j++) */
			/* 	B02 += 2*jn(j * 2 + 1, k*r2); */
                        B02 = k*r2*j0(k*r2)+M_PI*k*r2*(j1(k*r2)*STVH0(k*r2)-j0(k*r2)*STVH1(k*r2))/2;
                        B01 = k*r1*j0(k*r1)+M_PI*k*r1*(j1(k*r1)*STVH0(k*r1)-j0(k*r1)*STVH1(k*r1))/2;
			kernel += -b * (B02 - B01 ) /k / k / k;
		}
		out[indx] = kernel;
	}
}

__global__ void integral_Y(float *U_f, float *r, float *out, float *c, float *f, int nr, int nc, int nf) {
	bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k, fl, cl, r1, r2, g1, g2, a, b;
	float dr0, B01, B02;
	int indx_d;
	if ((gtidx >= nf) || (gtidy >= nc))
		validr = false;
	if (validr) {
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl / cl;
		kernel = 0.0;
		for (int i = 1; i < nr; i++) {
			indx_d = gtidx + i * nf;
			//g1 = U_f[indx_d-1];
			g1 = U_f[indx_d - nf];
			g2 = U_f[indx_d];
			r1 = r[i - 1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1), 0.1);
			a = g1 - r1 * (g2 - g1) / dr0;
			b = (g2 - g1) / dr0;
			kernel += a * (r2*y1(k*r2) - r1 * y1(k*r1)) / k;
			kernel += b * (r2*r2*y1(k*r2) - r1 * r1*y1(k*r1)) / k;
			kernel += b * (r2*y0(k*r2) - r1 * y0(k*r1)) / k / k;
			/* B01 = 0.0; */
			/* B0n = floorf(floorf(B01 / 2 + 10)*1.5); */
			/* for (int j = 0; j < B0n; j++) */
			/* 	B01 += 2*yn(j * 2 + 1, k*r1); */
			/* B02 = 0.0; */
			/* B0n = floorf(floorf(B02 / 2 + 10)*1.5); */
			/* for (int j = 0; j < B0n; j++) */
			/* 	B02 += 2*yn(j * 2 + 1, k*r2); */
                        B02 = k*r2*y0(k*r2)+M_PI*k*r2*(y1(k*r2)*STVH0(k*r2)-y0(k*r2)*STVH1(k*r2))/2;
                        B01 = k*r1*y0(k*r1)+M_PI*k*r1*(y1(k*r1)*STVH0(k*r1)-y0(k*r1)*STVH1(k*r1))/2;
			kernel += -b * (B02 - B01 ) /k / k / k;
		}
		out[indx] = kernel;
	}
}

__global__ void integral_c(float *U_f,float *r,float *out,float *c,float *f,int nr,int nc,int nf){
    bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k,fl,cl,r1,r2,g1,g2,a,b;
	float dr0,B01,B02;
	int indx_d,B0n;
	if ((gtidx >=nf) ||(gtidy>=nc))
		validr = false;
	if(validr){
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl/cl;
		kernel = 0.0;
		for(int i=1;i<nr;i++){
			indx_d = gtidx + i*nf;
			//g1 = U_f[indx_d-1];
			g1 = U_f[indx_d-nf];
			g2 = U_f[indx_d];
			r1 = r[i-1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1),0.1);
			a = g1 - r1*(g2-g1)/dr0;
			b = (g2-g1)/dr0;
			kernel += a*(r2*j1f(k*r2)-r1*j1f(k*r1))/k;
			kernel += b*(r2*r2*j1f(k*r2)-r1*r1*j1f(k*r1))/k;
			kernel += b*(r2*j0f(k*r2)-r1*j0f(k*r1))/k/k;
			B01 = 0.0;
			B0n = floorf(floorf(B01/2+10)*1.5);
			for(int j=0;j<B0n;j++)
				B01 += jnf(j*2+1,k*r1);
			B02 = 0.0;
			B0n = floorf(floorf(B02/2+10)*1.5);
			for(int j=0;j<B0n;j++)
				B02 += jnf(j*2+1,k*r2);
			//kernel += -b*(B02-B01+j0f(k*r1)*0.5+j0f(k*r2)*0.5)*dr/k/k;
			kernel += -b*(B02-B01)/k/k/k;
		}
		out[indx] = kernel;
	}
}

__global__ void trap_c(float *U_f,float *r,float *out,float *c,float *f,int nr,int nc,int nf){
	bool validr = true;
	const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
	int indx = 0;
	indx = gtidx + gtidy * nf;
	float kernel;
	float k,fl,cl,r1,r2,g1,g2,dr0;
	int indx_d;
	if ((gtidx >=nf) ||(gtidy>=nc))
		validr = false;
	if(validr){
		fl = f[gtidx];
		cl = c[gtidy];
		k = 2 * M_PI *fl/cl;
		kernel = 0.0;
		for(int i=1;i<nr;i++){
			indx_d = gtidx + i*nf;
			g1 = U_f[indx_d-nf];
			g2 = U_f[indx_d];
			r1 = r[i-1];
			r2 = r[i];
			dr0 = fmaxf((r2 - r1),0.1);
			kernel += 0.5*(j0f(k*r1)*g1*r1+j0f(k*r2)*g2*r2)*dr0;
		}
		out[indx] = kernel;
	}
}


int FJ(float *u_f,float *r, float *f,float *out, float *c,int nc,int nr,int nf,int type)
{	
    float *u_fc,*rc,*fc,*cc,*outc;
    dim3 block,grid;
    block.x = 32;
    block.y = 16;
	grid.x = (unsigned int)ceil((float)nf/block.x);
	grid.y = (unsigned int)ceil((float)nc/block.y);
    checkCudaErrors(hipMalloc((void **)&u_fc,nr*nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&rc,nr*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&fc,nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&cc,nc*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&outc,nf*nc*sizeof(float)));

    checkCudaErrors(hipMemcpy(u_fc,u_f,nr*nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(fc,f,nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rc,r,nr*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cc,c,nc*sizeof(float),hipMemcpyHostToDevice));
	if(type==1)
		integral_c<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
	if(type==0)
		trap_c<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
    checkCudaErrors(hipMemcpy(out, outc, nf*nc*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(u_fc));
    checkCudaErrors(hipFree(rc));
    checkCudaErrors(hipFree(fc));
    checkCudaErrors(hipFree(cc));
    checkCudaErrors(hipFree(outc));
    return 0;
}


int FHr(float *u_f,float *r, float *f,float *out, float *c,int nc,int nr,int nf,int type)
{
    float *u_fc,*rc,*fc,*cc,*outc;
    dim3 block,grid;
    block.x = 32;
    block.y = 16;
	grid.x = (unsigned int)ceil((float)nf/block.x);
	grid.y = (unsigned int)ceil((float)nc/block.y);
    checkCudaErrors(hipMalloc((void **)&u_fc,nr*nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&rc,nr*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&fc,nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&cc,nc*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&outc,nf*nc*sizeof(float)));

    checkCudaErrors(hipMemcpy(u_fc,u_f,nr*nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(fc,f,nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rc,r,nr*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cc,c,nc*sizeof(float),hipMemcpyHostToDevice));
	if(type==1)
		integral_J<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
	if(type==0)
		trap_J<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
    checkCudaErrors(hipMemcpy(out, outc, nf*nc*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(u_fc));
    checkCudaErrors(hipFree(rc));
    checkCudaErrors(hipFree(fc));
    checkCudaErrors(hipFree(cc));
    checkCudaErrors(hipFree(outc));
    return 0;
}


int FHi(float *u_f,float *r, float *f,float *out, float *c,int nc,int nr,int nf,int type)
{
    float *u_fc,*rc,*fc,*cc,*outc;
    dim3 block,grid;
    block.x = 32;
    block.y = 16;
	grid.x = (unsigned int)ceil((float)nf/block.x);
	grid.y = (unsigned int)ceil((float)nc/block.y);
    checkCudaErrors(hipMalloc((void **)&u_fc,nr*nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&rc,nr*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&fc,nf*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&cc,nc*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&outc,nf*nc*sizeof(float)));

    checkCudaErrors(hipMemcpy(u_fc,u_f,nr*nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(fc,f,nf*sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rc,r,nr*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cc,c,nc*sizeof(float),hipMemcpyHostToDevice));
	if(type==1)
		integral_Y<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
	if(type==0)
		trap_Y<<<grid,block>>>(u_fc,rc,outc,cc,fc,nr,nc,nf);
    checkCudaErrors(hipMemcpy(out, outc, nf*nc*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(u_fc));
    checkCudaErrors(hipFree(rc));
    checkCudaErrors(hipFree(fc));
    checkCudaErrors(hipFree(cc));
    checkCudaErrors(hipFree(outc));
    return 0;
}
